#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include "hiprand.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <memory>
#include <time.h>
#include <fstream>

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
#define checkCurandErrors(val) check_curand( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
	if (result)
	{
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		hipDeviceReset();
		exit(99);
	}
}

void check_curand(hiprandStatus_t result, char const *const func, const char *const file, int const line)
{
	if (result != HIPRAND_STATUS_SUCCESS)
	{
		std::cerr << "CURAND error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		hipDeviceReset();
		exit(101);
	}
}

__global__ void mulKernel(float *c, const float *a, const float *b, int n)
{
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	float result = 0.0f;
	if (row < n && col < n) {
		for (int i = 0; i < n; i++) {
			result += a[row * n + i] * b[i * n + col];
		}
		c[row * n + col] = result;
	}
	
}

void mulCPU(float *c, const float *a, const float *b, int n)
{
	for (int i = 0; i < n; ++i)
		for (int j = 0; j < n; ++j)
			c[n*j + i] = 0.0f;
	for (int i = 0; i < n; ++i)
		for (int j = 0; j < n; ++j)
			for (int k = 0; k < n; ++k)
				c[n*j + i] += a[k* n + i] * b[j * n + k];
}

void fill_matrix(float *a, int n)
{
	int size = n*n;
	hiprandGenerator_t gen;
	checkCurandErrors(hiprandCreateGeneratorHost(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
	checkCurandErrors(hiprandSetPseudoRandomGeneratorSeed(gen, clock()));
	checkCurandErrors(hiprandGenerateUniform(gen, a, size));

}

void print_matrix(float *c, int n)
{
	printf("\n");
	for (int i = 0; i < n; ++i)
	{
		for (int j = 0; j < n; ++j)
		{
			printf("%.2f ", c[n*j + i]);
		}
		printf("\n");
	}
}

void fill_test(float *a)
{
	float t[3][3] = { {1,2,3},{4,5,6},{7,8,9}};
	for (int i = 0; i < 3; ++i)
		for (int j = 0; j < 3; ++j)
			a[j*3 + i] = t[i][j];
}

int main()
{
	hipDeviceProp_t prop;
	checkCudaErrors(hipGetDeviceProperties(&prop, 0));
	printf("Your CUDA-compatible device: %s\n", prop.name);

	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	std::ofstream out("out.txt");

	for (int n = 200; n < 1600; n += 200)
	{
		int size = n * n * sizeof(float);
		float *a = (float*)malloc(size);
		float *b = (float*)malloc(size);
		float *c = (float*)malloc(size);

		//fill_test(a);
		//fill_test(b);
		fill_matrix(a, n);
		fill_matrix(b, n);
		fill_matrix(c, n);

		float *a_gpu, *b_gpu, *c_gpu;

		checkCudaErrors(hipMalloc((void**)&a_gpu, size));
		checkCudaErrors(hipMalloc((void**)&b_gpu, size));
		checkCudaErrors(hipMalloc((void**)&c_gpu, size));

		checkCudaErrors(hipMemcpy(a_gpu, a, size, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(b_gpu, b, size, hipMemcpyHostToDevice));

		checkCudaErrors(hipEventRecord(start));

		int tx = 16, ty = 16;
		dim3 blocks(n / tx + 1, n / ty + 1);
		dim3 threads(tx, ty);

		mulKernel << <blocks, threads >> > (c_gpu, a_gpu, b_gpu, n);
		checkCudaErrors(hipEventRecord(stop));
		checkCudaErrors(hipEventSynchronize(stop));
		hipDeviceSynchronize();

		float milliseconds = 0;
		checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));

		milliseconds /= 1000.0f;

		printf("\nGPU calculating time: %.5f", milliseconds);

		checkCudaErrors(hipMemcpy(c, c_gpu, size, hipMemcpyDeviceToHost));

		clock_t start1 = clock();
		mulCPU(c, a, b, n);

		float elapsed = difftime(clock(), start1) / CLOCKS_PER_SEC;

		printf("\nCPU calculating time: %.5f", elapsed);

		printf("\nSpeedup: %.2f\n", elapsed / milliseconds);

		out << n << " " << milliseconds << " " << elapsed << " " << (elapsed / milliseconds) << std::endl;

		free(a);
		free(b);
		free(c);

		checkCudaErrors(hipFree(a_gpu));
		checkCudaErrors(hipFree(b_gpu));
		checkCudaErrors(hipFree(c_gpu));
	}
	out.close();
	return 0;
}